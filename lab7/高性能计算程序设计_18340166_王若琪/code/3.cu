/******************************************
 * Compile:
 * nvcc 3.cu -o 3 -I/opt/conda/include -L/opt/conda/lib -lcudnn
 * Run:       
 * ./3
*******************************************/

#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define input_height 4096
#define input_width 4096
#define filter_size 3
#define stride 3
#define pad 1
#define channels 3
#define output_height (input_height + 2*pad - filter_size) / stride + 1
#define output_width (input_width + 2*pad - filter_size) / stride + 1

#define checkCUDNN(expression)                                     \
    {                                                              \
        hipdnnStatus_t status = (expression);                       \
        if (status != HIPDNN_STATUS_SUCCESS)                        \
        {                                                          \
            std::cerr << "Error on line " << __LINE__ << ": "      \
                      << hipdnnGetErrorString(status) << std::endl; \
            std::exit(EXIT_FAILURE);                               \
        }                                                          \
    }

int main(int argc, char const *argv[])
{
    hipdnnHandle_t cudnn;
	hipdnnCreate(&cudnn);
	
	float *image = (float *)malloc(input_height * input_width * channels * sizeof(float));
	// 初始化input矩阵
	for (int i = 0; i < input_height * input_width * channels ; i++)
	{
		image[i] = (float)(rand() % 50)/100;
	}


    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/3,
                                          /*image_height=*/input_height,
                                          /*image_width=*/input_width));

    //output descriptor
    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(output_descriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/1,
                                          /*image_height=*/output_height,
                                          /*image_width=*/output_width));

    //descriptor kernel
    hipdnnFilterDescriptor_t kernel_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/1,
                                          /*in_channels=*/3,
                                          /*kernel_height=*/3,
                                          /*kernel_width=*/3));

    //descriptor convolucion
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                               /*pad_height=*/pad,
                                               /*pad_width=*/pad,
                                               /*vertical_stride=*/stride,
                                               /*horizontal_stride=*/stride,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION,
                                               /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(
        hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                            input_descriptor,
                                            kernel_descriptor,
                                            convolution_descriptor,
                                            output_descriptor,
                                            HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                            /*memoryLimitInBytes=*/0,
                                            &convolution_algorithm));

    size_t workspace_bytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       kernel_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));

    //分配空间
    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);
    int image_bytes = 1 * 3 * input_height * input_width * sizeof(float);
    float *d_input{nullptr};
    hipMalloc(&d_input, image_bytes);
    hipMemcpy(d_input, image, image_bytes, hipMemcpyHostToDevice);
    float *d_output{nullptr};
    hipMalloc(&d_output, image_bytes);
    hipMemset(d_output, 0, image_bytes);

    // 初始化 kernel
    float h_kernel[3][1][3][3];
    for (int kernel = 0; kernel < 3; ++kernel)
    {
        for (int channel = 0; channel < 1; ++channel)
        {
            for (int row = 0; row < 3; ++row)
            {
                for (int column = 0; column < 3; ++column)
                {
                    h_kernel[kernel][channel][row][column] = (float)(rand() % 50)/100;
                }
            }
        }
    }

    float *d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);

    const float alpha = 1, beta = 1;

    // convolution forward
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       d_input,
                                       kernel_descriptor,
                                       d_kernel,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       d_output));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time : %f\n", milliseconds / (3.0f * 1000.0f));
    
    float *h_output = (float *)malloc(image_bytes);
	hipMemcpy(h_output, d_output, image_bytes, hipMemcpyDeviceToHost);
    
    // 输出结果到文件result.txt
    FILE *fp = fopen("result.txt", "w");
    for (int i = 0; i < output_width * output_height; i++)
    {
        fprintf(fp, "%f ", h_output[i]);
    }
    
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    free(image);
	free(h_output);
	free(d_workspace);
    
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroyFilterDescriptor(kernel_descriptor);

    hipdnnDestroy(cudnn);
    
    return 0;
}